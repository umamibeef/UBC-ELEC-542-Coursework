#include "hip/hip_runtime.h"
/** 
MIT License

Copyright (c) [2022] [Michel Kakulphimp]

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
**/

// C/C++ includes
#include <stdlib.h>
#include <stdio.h>
#include <chrono>

// Boost includes
#include <boost/format.hpp>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hipsolver.h>

// Program includes
#include "main.hpp"
#include "config.hpp"
#include "console.hpp"
#include "kernel.h"

using namespace boost;

// Derived from:
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/utils/cusolver_utils.h
// CUDA API error checking
#define CUDA_CHECK(err)                                                                                                 \
    do                                                                                                                  \
    {                                                                                                                   \
        hipError_t err_ = (err);                                                                                       \
        if (err_ != hipSuccess)                                                                                        \
        {                                                                                                               \
            console_print_err(0, str(format("CUDA error %d at %s:%d\n") % err_ % __FILE__ % __LINE__), CLIENT_CUDA);    \
            throw std::runtime_error("CUDA error");                                                                     \
        }                                                                                                               \
    }                                                                                                                   \
    while (0)                                                                                                           \

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                                                             \
    do                                                                                                                  \
    {                                                                                                                   \
        hipsolverStatus_t err_ = (err);                                                                                  \
        if (err_ != HIPSOLVER_STATUS_SUCCESS)                                                                            \
        {                                                                                                               \
            console_print_err(0, str(format("cusolver error %d at %s:%d\n") % err_ % __FILE__ % __LINE__), CLIENT_CUSOLVER);\
            throw std::runtime_error("cusolver error");                                                                 \
        }                                                                                                               \
    }                                                                                                                   \
    while (0)                                                                                                           \

int multi_processor_count;
int max_blocks_per_multiprocessor;
int max_threads_per_multiprocessor;

// Electron-electron Coulombic repulsion function
__device__
float cuda_repulsion_function(Lut_t lut, int linear_coordinates_1, int linear_coordinates_2)
{
    const float epsilon = EPSILON;

    float x1 = lut.coordinate_value_array[IDX_X * lut.matrix_dim + linear_coordinates_1];
    float y1 = lut.coordinate_value_array[IDX_Y * lut.matrix_dim + linear_coordinates_1];
    float z1 = lut.coordinate_value_array[IDX_Z * lut.matrix_dim + linear_coordinates_1];

    float x2 = lut.coordinate_value_array[IDX_X * lut.matrix_dim + linear_coordinates_2];
    float y2 = lut.coordinate_value_array[IDX_Y * lut.matrix_dim + linear_coordinates_2];
    float z2 = lut.coordinate_value_array[IDX_Z * lut.matrix_dim + linear_coordinates_2];

    float denominator = sqrtf((x2 - x1)*(x2 - x1) + (y2 - y1)*(y2 - y1) + (z2 - z1)*(z2 - z1));

    if (abs(denominator) < epsilon)
    {
        denominator = sqrtf(TINY_NUMBER);
    }

    return (1.0/(denominator));
}

__device__
float cuda_repulsion_matrix_integrand_function(Lut_t lut, float *orbital_values, int linear_coords_1, int linear_coords_2)
{
    return orbital_values[linear_coords_2] * orbital_values[linear_coords_2] * cuda_repulsion_function(lut, linear_coords_1, linear_coords_2);
}

__device__
float cuda_exchange_matrix_integrand_function(Lut_t lut, float *orbital_values, int linear_coords_1, int linear_coords_2)
{
    return orbital_values[linear_coords_1] * orbital_values[linear_coords_2] * cuda_repulsion_function(lut, linear_coords_1, linear_coords_2);
}

__global__
void cuda_generate_repulsion_matrix_kernel(Lut_t lut, float *orbital_values, float *repulsion_diagonal)
{
    int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int electron_one_coordinate_index = start_index; electron_one_coordinate_index < lut.matrix_dim; electron_one_coordinate_index += stride)
    {
        float sum = 0;
        for (int electron_two_coordinate_index = 0; electron_two_coordinate_index < lut.matrix_dim; electron_two_coordinate_index++)
        {
            sum += cuda_repulsion_matrix_integrand_function(lut, orbital_values, electron_one_coordinate_index, electron_two_coordinate_index);
        }
        repulsion_diagonal[electron_one_coordinate_index] = sum * lut.step_size_cubed;
    }
}

__global__
void cuda_generate_exchange_matrix_kernel(Lut_t lut, float *orbital_values, float *exchange_diagonal)
{
    int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int electron_one_coordinate_index = start_index; electron_one_coordinate_index < lut.matrix_dim; electron_one_coordinate_index += stride)
    {
        float sum = 0;
        for (int electron_two_coordinate_index = 0; electron_two_coordinate_index < lut.matrix_dim; electron_two_coordinate_index++)
        {
            sum += cuda_exchange_matrix_integrand_function(lut, orbital_values, electron_one_coordinate_index, electron_two_coordinate_index);
        }
        exchange_diagonal[electron_one_coordinate_index] = sum * lut.step_size_cubed;
    }
}

void cuda_print_memory_info()
{
    size_t free;
    size_t total;

    hipMemGetInfo(&free,&total);
    console_print(2, str(format("Memory Free: %zu Memory Total: %zu") % free % total), CLIENT_CUDA);
}

void cuda_device_reset(void)
{
    CUDA_CHECK(hipDeviceReset()); // Reset CUDA device
}

int cuda_get_device_info(void)
{
    int num_devices;

    // cuda reset
    cuda_device_reset();

    console_print_hr(0, CLIENT_CUDA);
    console_print(0, "CUDA Device information:", CLIENT_CUDA);
    hipGetDeviceCount(&num_devices);
    for (int i = 0; i < num_devices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        console_print(0, str(format("Device Number: %d\n") % i), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Device name: %s\n") % prop.name), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "CUDA Capability %d.%d\n") % prop.major % prop.minor), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Memory Clock Rate (kHz): %d\n") % prop.memoryClockRate), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Memory Bus Width (bits): %d\n") % prop.memoryBusWidth), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Peak Memory Bandwidth (GB/s): %f\n") % (2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6)), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Streaming Multiprocessors Count: %d\n") % prop.multiProcessorCount), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Blocks per Streaming Multiprocessors: %d\n") % prop.maxBlocksPerMultiProcessor), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Threads per Streaming Multiprocessors: %d\n") % prop.maxThreadsPerMultiProcessor), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Dimensions of a Thread Block (x,y,z): (%d, %d, %d)\n") % prop.maxThreadsDim[0] % prop.maxThreadsDim[1] % prop.maxThreadsDim[2]), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Dimensions of a Grid Size (x,y,z): (%d, %d, %d)\n") % prop.maxGridSize[0] % prop.maxGridSize[1] % prop.maxGridSize[2]), CLIENT_CUDA);

        // TODO: handle the multi CUDA device case
        multi_processor_count = prop.multiProcessorCount;
        max_blocks_per_multiprocessor = prop.maxBlocksPerMultiProcessor;
        max_threads_per_multiprocessor = prop.maxThreadsPerMultiProcessor;
    }
    cuda_print_memory_info();
    if (num_devices == 0)
    {
        console_print_warn(0, "No CUDA devices available!", CLIENT_CUDA);
    }

    return num_devices;
}

int cuda_allocate_integration_memory(Lut_t &lut, DynamicDataPointers_t &ddp)
{
    int rv = 0;
    hipError_t error;

    console_print_hr(2, CLIENT_CUDA);
    console_print(2, "Allocating memory for CUDA integration...", CLIENT_CUDA);

    size_t orbital_vector_size_bytes = sizeof(float) * lut.matrix_dim;
    size_t repulsion_exchange_matrices_size_bytes = sizeof(float) * lut.matrix_dim;
    size_t coordinate_luts_size_bytes = sizeof(float) * IDX_NUM * lut.matrix_dim;

    hipMalloc(&(ddp.orbital_values_data), orbital_vector_size_bytes);
    hipMalloc(&(ddp.repulsion_diagonal_data), repulsion_exchange_matrices_size_bytes);
    hipMalloc(&(ddp.exchange_diagonal_data), repulsion_exchange_matrices_size_bytes);

    hipMalloc(&(lut.coordinate_value_array), coordinate_luts_size_bytes);
    hipMalloc(&(lut.coordinate_index_array), coordinate_luts_size_bytes);

    console_print(2, str(format("Trying to allocate %zu bytes for orbital values vector") % orbital_vector_size_bytes), CLIENT_CUDA);
    console_print(2, str(format("Trying to allocate %zu bytes for repulsion matrix diagonal") % repulsion_exchange_matrices_size_bytes), CLIENT_CUDA);
    console_print(2, str(format("Trying to allocate %zu bytes for exchange matrix diagonal") % repulsion_exchange_matrices_size_bytes), CLIENT_CUDA);
    console_print(2, str(format("Trying to allocate 3x %zu bytes for coordinate LUTs") % coordinate_luts_size_bytes), CLIENT_CUDA);

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, str(format("Allocated %zu bytes for orbital values vector") % orbital_vector_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated %zu bytes for repulsion matrix diagonal") % repulsion_exchange_matrices_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated %zu bytes for exchange matrix diagonal") % repulsion_exchange_matrices_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated 2x %zu bytes for coordinate LUTs") % coordinate_luts_size_bytes), CLIENT_CUDA);
        cuda_print_memory_info();
    }

    return rv;
}

int cuda_allocate_eigensolver_memory(Lut_t &lut, DynamicDataPointers_t &ddp)
{
    int rv = 0;
    hipError_t error;

    console_print_hr(2, CLIENT_CUDA);
    console_print(2, "Allocating memory for CUDA eigensolver...", CLIENT_CUDA);

    size_t eigenvectors_data_size_bytes = sizeof(float) * lut.matrix_dim * lut.matrix_dim;
    size_t eigenvalues_data_size_bytes = sizeof(float) * lut.matrix_dim;

    console_print(2, str(format("Trying to allocate %zu bytes for eigenvectors matrix") % eigenvectors_data_size_bytes), CLIENT_CUDA);
    console_print(2, str(format("Trying to allocate %zu bytes for eigenvalues vector") % eigenvalues_data_size_bytes), CLIENT_CUDA);

    hipMalloc(&(ddp.eigenvectors_data), eigenvectors_data_size_bytes);
    hipMalloc(&(ddp.eigenvalues_data), eigenvalues_data_size_bytes);

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, str(format("Allocated %zu bytes for eigenvectors matrix") % eigenvectors_data_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated %zu bytes for eigenvalues vector") % eigenvalues_data_size_bytes), CLIENT_CUDA);
        cuda_print_memory_info();
    }

    return rv;
}

int cuda_free_integration_memory(Lut_t &lut, DynamicDataPointers_t &ddp)
{
    int rv = 0;
    hipError_t error;

    console_print(2, "Freeing allocated integration memory...", CLIENT_CUDA);

    hipFree((void*)ddp.orbital_values_data);
    hipFree((void*)ddp.repulsion_diagonal_data);
    hipFree((void*)ddp.exchange_diagonal_data);
    hipFree((void*)lut.coordinate_value_array);
    hipFree((void*)lut.coordinate_index_array);

    // null the pointers
    ddp.orbital_values_data = nullptr;
    ddp.repulsion_diagonal_data = nullptr;
    ddp.exchange_diagonal_data = nullptr;
    lut.coordinate_value_array = nullptr;
    lut.coordinate_index_array = nullptr;

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, "Successfully freed allocated integration memory", CLIENT_CUDA);
        cuda_print_memory_info();
    }

    return rv;
}

int cuda_free_eigensolver_memory(DynamicDataPointers_t &ddp)
{
    int rv = 0;
    hipError_t error;

    console_print(2, "Freeing allocated eigensolver memory...", CLIENT_CUDA);

    hipFree((void*)ddp.eigenvectors_data);
    hipFree((void*)ddp.eigenvalues_data);

    // null the pointer
    ddp.eigenvectors_data = nullptr;
    ddp.eigenvalues_data = nullptr;

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, "Successfully freed allocated eigensolver memory", CLIENT_CUDA);
        cuda_print_memory_info();
    }

    return rv;
}

void cuda_numerical_integration(Lut_t lut, DynamicDataPointers_t ddp)
{
    Lut_t cuda_lut = lut;
    DynamicDataPointers_t cuda_ddp;

    int num_blocks = multi_processor_count * max_blocks_per_multiprocessor;
    int blocks_size = multi_processor_count * max_threads_per_multiprocessor / num_blocks;

    // cuda reset
    cuda_device_reset();

    // allocate memory
    cuda_allocate_integration_memory(cuda_lut, cuda_ddp);

    size_t orbital_vector_size_bytes = sizeof(float) * lut.matrix_dim;
    size_t repulsion_exchange_matrices_size_bytes = sizeof(float) * lut.matrix_dim;
    size_t coordinate_luts_size_bytes = sizeof(float) * IDX_NUM * lut.matrix_dim;

    // copy data to device
    CUDA_CHECK(hipMemcpy(cuda_ddp.orbital_values_data, ddp.orbital_values_data, orbital_vector_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_ddp.repulsion_diagonal_data, ddp.repulsion_diagonal_data, repulsion_exchange_matrices_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_ddp.exchange_diagonal_data, ddp.exchange_diagonal_data, repulsion_exchange_matrices_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_lut.coordinate_value_array, lut.coordinate_value_array, coordinate_luts_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_lut.coordinate_index_array, lut.coordinate_index_array, coordinate_luts_size_bytes, hipMemcpyHostToDevice));

    console_print(0, "Computing repulsion matrix", CLIENT_CUDA);
    cuda_generate_repulsion_matrix_kernel<<<num_blocks, blocks_size>>>(cuda_lut, cuda_ddp.orbital_values_data, cuda_ddp.repulsion_diagonal_data);
    CUDA_CHECK(hipDeviceSynchronize());

    console_print(0, "Computing exchange matrix", CLIENT_CUDA);
    cuda_generate_exchange_matrix_kernel<<<num_blocks, blocks_size>>>(cuda_lut, cuda_ddp.orbital_values_data, cuda_ddp.exchange_diagonal_data);
    CUDA_CHECK(hipDeviceSynchronize());

    // copy results to host
    CUDA_CHECK(hipMemcpy(ddp.repulsion_diagonal_data, cuda_ddp.repulsion_diagonal_data, repulsion_exchange_matrices_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(ddp.exchange_diagonal_data, cuda_ddp.exchange_diagonal_data, repulsion_exchange_matrices_size_bytes, hipMemcpyDeviceToHost));

    // free memory
    cuda_free_integration_memory(cuda_lut, cuda_ddp); 
}

bool cuda_eigensolver(Lut_t lut, DynamicDataPointers_t ddp)
{
    hipsolverHandle_t cusolver_handle = NULL;
    hipStream_t stream = NULL;
    int *device_info_ptr = nullptr; // pointer to device algorithm info in
    int info = 0; // algorithm info int
    int lwork = 0; // size of workspace
    float *workspace_ptr = nullptr; // pointer to workspace
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // solver job type, compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER; // matrix fill mode;
    DynamicDataPointers_t cuda_ddp;

    size_t eigenvectors_data_size_bytes = sizeof(float) * lut.matrix_dim * lut.matrix_dim;
    size_t eigenvalues_data_size_bytes = sizeof(float) * lut.matrix_dim;

    // cuda reset
    cuda_device_reset();

    // allocate memory solution
    cuda_allocate_eigensolver_memory(lut, cuda_ddp);

    // copy data to device
    CUDA_CHECK(hipMemcpy(cuda_ddp.eigenvectors_data, ddp.eigenvectors_data, eigenvectors_data_size_bytes, hipMemcpyHostToDevice));

    console_print(0, "hipsolverDnSsyevd start", CLIENT_CUSOLVER);
    console_print(2, TAB1 "hipsolverDnSsyevd solver debug", CLIENT_CUSOLVER);

    // create a cusolver handle and bind it to a stream
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolver_handle, stream));

    // allocate a spot for device_info_ptr
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&device_info_ptr), sizeof(int)));

    // query working space required for syevd
    console_print(2, TAB2 "hipsolverDnSsyevd_bufferSize query", CLIENT_CUSOLVER);
    CUSOLVER_CHECK(hipsolverDnSsyevd_bufferSize(cusolver_handle, jobz, uplo, lut.matrix_dim, cuda_ddp.eigenvectors_data, lut.matrix_dim, cuda_ddp.eigenvalues_data, &lwork));
    
    console_print(2, str(format(TAB2"lwork = %d") % lwork), CLIENT_CUSOLVER);

    // allocate memory for work area
    console_print(2, TAB2 "allocating workspace", CLIENT_CUSOLVER);
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&workspace_ptr), sizeof(float) * lwork));

    // compute solution
    console_print(2, TAB2 "calling hipsolverDnSsyevd", CLIENT_CUSOLVER);
    CUSOLVER_CHECK(hipsolverDnSsyevd(cusolver_handle, jobz, uplo, lut.matrix_dim, cuda_ddp.eigenvectors_data, lut.matrix_dim, cuda_ddp.eigenvalues_data, workspace_ptr, lwork, device_info_ptr));
    CUDA_CHECK(hipMemcpyAsync(&info, device_info_ptr, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    console_print(2, str(format(TAB2"info = %d") % info), CLIENT_CUSOLVER);

    // copy results to host
    CUDA_CHECK(hipMemcpy(ddp.eigenvectors_data, cuda_ddp.eigenvectors_data, eigenvectors_data_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(ddp.eigenvalues_data, cuda_ddp.eigenvalues_data, eigenvalues_data_size_bytes, hipMemcpyDeviceToHost));

    // free memory
    cuda_free_eigensolver_memory(cuda_ddp);

    return (info==0);
}