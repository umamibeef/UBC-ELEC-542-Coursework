#include "hip/hip_runtime.h"
/** 
MIT License

Copyright (c) [2022] [Michel Kakulphimp]

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
**/

// C/C++ includes
#include <stdlib.h>
#include <stdio.h>
#include <chrono>

// Boost includes
#include <boost/format.hpp>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hipsolver.h>

// Program includes
#include "main.hpp"
#include "config.hpp"
#include "console.hpp"
#include "kernel.h"

using namespace boost;

// Derived from:
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/utils/cusolver_utils.h
// CUDA API error checking
#define CUDA_CHECK(err)                                                                                                 \
    do                                                                                                                  \
    {                                                                                                                   \
        hipError_t err_ = (err);                                                                                       \
        if (err_ != hipSuccess)                                                                                        \
        {                                                                                                               \
            console_print_err(0, str(format("CUDA error %d at %s:%d\n") % err_ % __FILE__ % __LINE__), CLIENT_CUDA);    \
            throw std::runtime_error("CUDA error");                                                                     \
        }                                                                                                               \
    }                                                                                                                   \
    while (0)                                                                                                           \

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                                                             \
    do                                                                                                                  \
    {                                                                                                                   \
        hipsolverStatus_t err_ = (err);                                                                                  \
        if (err_ != HIPSOLVER_STATUS_SUCCESS)                                                                            \
        {                                                                                                               \
            console_print_err(0, str(format("cusolver error %d at %s:%d\n") % err_ % __FILE__ % __LINE__), CLIENT_CUDA);\
            throw std::runtime_error("cusolver error");                                                                 \
        }                                                                                                               \
    }                                                                                                                   \
    while (0)                                                                                                           \

int multi_processor_count;
int max_blocks_per_multiprocessor;
int max_threads_per_multiprocessor;

// Electron-electron Coulombic repulsion function
__device__
float cuda_repulsion_function(LutVals_t lut_vals, int linear_coordinates_1, int linear_coordinates_2)
{
    const float epsilon = EPSILON;

    float x1 = lut_vals.coordinate_value_array[IDX_X * lut_vals.matrix_dim + linear_coordinates_1];
    float y1 = lut_vals.coordinate_value_array[IDX_Y * lut_vals.matrix_dim + linear_coordinates_1];
    float z1 = lut_vals.coordinate_value_array[IDX_Z * lut_vals.matrix_dim + linear_coordinates_1];

    float x2 = lut_vals.coordinate_value_array[IDX_X * lut_vals.matrix_dim + linear_coordinates_2];
    float y2 = lut_vals.coordinate_value_array[IDX_Y * lut_vals.matrix_dim + linear_coordinates_2];
    float z2 = lut_vals.coordinate_value_array[IDX_Z * lut_vals.matrix_dim + linear_coordinates_2];

    float denominator = sqrtf((x2 - x1)*(x2 - x1) + (y2 - y1)*(y2 - y1) + (z2 - z1)*(z2 - z1));

    if (abs(denominator) < epsilon)
    {
        denominator = sqrtf(TINY_NUMBER);
    }

    return (1.0/(denominator));
}

__device__
float cuda_repulsion_matrix_integrand_function(LutVals_t lut_vals, float *orbital_values, int linear_coords_1, int linear_coords_2)
{
    return orbital_values[linear_coords_2] * orbital_values[linear_coords_2] * cuda_repulsion_function(lut_vals, linear_coords_1, linear_coords_2);
}

__device__
float cuda_exchange_matrix_integrand_function(LutVals_t lut_vals, float *orbital_values, int linear_coords_1, int linear_coords_2)
{
    return orbital_values[linear_coords_1] * orbital_values[linear_coords_2] * cuda_repulsion_function(lut_vals, linear_coords_1, linear_coords_2);
}

__global__
void cuda_generate_repulsion_matrix_kernel(LutVals_t lut_vals, float *orbital_values, float *repulsion_diagonal)
{
    int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int electron_one_coordinate_index = start_index; electron_one_coordinate_index < lut_vals.matrix_dim; electron_one_coordinate_index += stride)
    {
        float sum = 0;
        for (int electron_two_coordinate_index = 0; electron_two_coordinate_index < lut_vals.matrix_dim; electron_two_coordinate_index++)
        {
            sum += cuda_repulsion_matrix_integrand_function(lut_vals, orbital_values, electron_one_coordinate_index, electron_two_coordinate_index);
        }
        repulsion_diagonal[electron_one_coordinate_index] = sum * lut_vals.step_size_cubed;
    }
}

__global__
void cuda_generate_exchange_matrix_kernel(LutVals_t lut_vals, float *orbital_values, float *exchange_diagonal)
{
    int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int electron_one_coordinate_index = start_index; electron_one_coordinate_index < lut_vals.matrix_dim; electron_one_coordinate_index += stride)
    {
        float sum = 0;
        for (int electron_two_coordinate_index = 0; electron_two_coordinate_index < lut_vals.matrix_dim; electron_two_coordinate_index++)
        {
            sum += cuda_exchange_matrix_integrand_function(lut_vals, orbital_values, electron_one_coordinate_index, electron_two_coordinate_index);
        }
        exchange_diagonal[electron_one_coordinate_index] = sum * lut_vals.step_size_cubed;
    }
}

int cuda_get_device_info(void)
{
    int num_devices;

    console_print_spacer(0, CLIENT_CUDA);
    console_print(0, "CUDA Device information:", CLIENT_CUDA);
    hipGetDeviceCount(&num_devices);
    for (int i = 0; i < num_devices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        console_print(0, str(format("Device Number: %d\n") % i), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Device name: %s\n") % prop.name), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "CUDA Capability %d.%d\n") % prop.major % prop.minor), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Memory Clock Rate (kHz): %d\n") % prop.memoryClockRate), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Memory Bus Width (bits): %d\n") % prop.memoryBusWidth), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Peak Memory Bandwidth (GB/s): %f\n") % (2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6)), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Streaming Multiprocessors Count: %d\n") % prop.multiProcessorCount), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Blocks per Streaming Multiprocessors: %d\n") % prop.maxBlocksPerMultiProcessor), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Threads per Streaming Multiprocessors: %d\n") % prop.maxThreadsPerMultiProcessor), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Dimensions of a Thread Block (x,y,z): (%d, %d, %d)\n") % prop.maxThreadsDim[0] % prop.maxThreadsDim[1] % prop.maxThreadsDim[2]), CLIENT_CUDA);
        console_print(0, str(format(TAB1 "Max Dimensions of a Grid Size (x,y,z): (%d, %d, %d)\n") % prop.maxGridSize[0] % prop.maxGridSize[1] % prop.maxGridSize[2]), CLIENT_CUDA);

        // TODO: handle the multi CUDA device case
        multi_processor_count = prop.multiProcessorCount;
        max_blocks_per_multiprocessor = prop.maxBlocksPerMultiProcessor;
        max_threads_per_multiprocessor = prop.maxThreadsPerMultiProcessor;

    }
    if (num_devices == 0)
    {
        console_print_warn(0, "No CUDA devices available!", CLIENT_CUDA);
    }
    console_print_spacer(0, CLIENT_CUDA);

    return num_devices;
}

int cuda_allocate_integration_memory(LutVals_t *lut_vals, float **orbital_values_data, float **repulsion_diagonal_data, float **exchange_diagonal_data)
{
    int rv = 0;
    hipError_t error;

    console_print(0, "Allocating memory for CUDA integration...", CLIENT_CUDA);

    int orbital_vector_size_bytes = lut_vals->matrix_dim * sizeof(float);
    int repulsion_exchange_matrices_size_bytes = lut_vals->matrix_dim * sizeof(float);
    int coordinate_luts_size_bytes = IDX_NUM * lut_vals->matrix_dim * sizeof(float);

    hipMallocManaged(orbital_values_data, orbital_vector_size_bytes);
    hipMallocManaged(repulsion_diagonal_data, repulsion_exchange_matrices_size_bytes);
    hipMallocManaged(exchange_diagonal_data, repulsion_exchange_matrices_size_bytes);

    hipMallocManaged(&(lut_vals->coordinate_value_array), coordinate_luts_size_bytes);
    hipMallocManaged(&(lut_vals->coordinate_index_array), coordinate_luts_size_bytes);

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, str(format("Allocated %d bytes for orbital values vector") % orbital_vector_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated %d bytes for repulsion matrix diagonal") % repulsion_exchange_matrices_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated %d bytes for exchange matrix diagonal") % repulsion_exchange_matrices_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated 3x %d bytes for coordinate LUTs") % coordinate_luts_size_bytes), CLIENT_CUDA);
    }
    console_print_spacer(0, CLIENT_CUDA);

    return rv;
}

int cuda_allocate_eigensolver_memory(LutVals_t *lut_vals, float **eigenvectors_data, float **eigenvalues_data)
{
    int rv = 0;
    hipError_t error;

    console_print(0, "Allocating memory for CUDA eigensolver...", CLIENT_CUDA);

    int eigenvectors_data_size_bytes = lut_vals->matrix_dim * lut_vals->matrix_dim * sizeof(float);
    int eigenvalues_data_size_bytes = lut_vals->matrix_dim * sizeof(float);

    hipMallocManaged(eigenvectors_data, eigenvectors_data_size_bytes);
    hipMallocManaged(eigenvalues_data, eigenvalues_data_size_bytes);

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, str(format("Allocated %d bytes for eigenvectors matrix") % eigenvectors_data_size_bytes), CLIENT_CUDA);
        console_print(2, str(format("Allocated %d bytes for eigenvalues vector") % eigenvalues_data_size_bytes), CLIENT_CUDA);
    }
    console_print_spacer(0, CLIENT_CUDA);

    return rv;
}

int cuda_free_integration_memory(LutVals_t *lut_vals, float **orbital_values_data, float **repulsion_diagonal_data, float **exchange_diagonal_data)
{
    int rv = 0;
    hipError_t error;

    console_print(0, "Freeing allocated integration memory...", CLIENT_CUDA);

    hipFree(*orbital_values_data);
    hipFree(*repulsion_diagonal_data);
    hipFree(*exchange_diagonal_data);
    hipFree(lut_vals->coordinate_value_array);
    hipFree(lut_vals->coordinate_index_array);

    // null the pointers
    (*orbital_values_data) = nullptr;
    (*repulsion_diagonal_data) = nullptr;
    (*exchange_diagonal_data) = nullptr;
    (lut_vals->coordinate_value_array) = nullptr;
    (lut_vals->coordinate_index_array) = nullptr;

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, "Successfully freed allocated integration memory", CLIENT_CUDA);
    }

    return rv;
}

int cuda_free_eigensolver_memory(float **eigenvectors_data, float **eigenvalues_data)
{
    int rv = 0;
    hipError_t error;

    console_print(0, "Freeing allocated eigensolver memory...", CLIENT_CUDA);

    hipFree(*eigenvectors_data);
    hipFree(*eigenvalues_data);

    // null the pointer
    (*eigenvectors_data) = nullptr;
    (*eigenvalues_data) = nullptr;

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }
    else
    {
        console_print(2, "Successfully freed allocated eigensolver memory", CLIENT_CUDA);
    }

    return rv;
}

int cuda_numerical_integration(LutVals_t lut_vals, float *orbital_values, float *repulsion_matrix, float *exchange_matrix)
{
    int rv = 0;
    hipError_t error;

    int num_blocks = multi_processor_count * max_blocks_per_multiprocessor;
    int blocks_size = multi_processor_count * max_threads_per_multiprocessor / num_blocks;

    console_print(0, "Computing repulsion matrix", CLIENT_CUDA);
    cuda_generate_repulsion_matrix_kernel<<<num_blocks, blocks_size>>>(lut_vals, orbital_values, repulsion_matrix);
    hipDeviceSynchronize();

    console_print(0, "Computing exchange matrix", CLIENT_CUDA);
    cuda_generate_exchange_matrix_kernel<<<num_blocks, blocks_size>>>(lut_vals, orbital_values, exchange_matrix);
    hipDeviceSynchronize();

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        console_print_err(0, str(format("%s\n") % hipGetErrorString(error)), CLIENT_CUDA);
        rv = 1;
    }

    return rv;
}

bool cuda_eigensolver(LutVals_t lut_vals, float *eigenvectors_data, float *eigenvalues_data)
{
    hipError_t cuda_error;
    hipsolverStatus_t cusolver_error;
    hipsolverHandle_t cusolver_handle = NULL;
    hipStream_t stream = NULL;
    int *device_info_ptr = nullptr; // pointer to device algorithm info in
    int info = 0; // algorithm info int
    int lwork = 0; // size of workspace
    float *workspace_ptr = nullptr; // pointer to workspace
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // solver job type, compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER; // matrix fill mode;

    // Create a cusolver handle and bind it to a stream
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolver_handle, stream));

    // Allocate a spot for device_info_ptr
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&device_info_ptr), sizeof(int)));

    // Query working space required for syevd
    CUSOLVER_CHECK(hipsolverDnSsyevd_bufferSize(cusolver_handle, jobz, uplo, lut_vals.matrix_dim, eigenvectors_data, lut_vals.matrix_dim, eigenvalues_data, &lwork));
    
    // Allocate memory for work area
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&workspace_ptr), sizeof(float) * lwork));

    // compute solution
    CUSOLVER_CHECK(hipsolverDnSsyevd(cusolver_handle, jobz, uplo, lut_vals.matrix_dim, eigenvectors_data, lut_vals.matrix_dim, eigenvalues_data, workspace_ptr, lwork, device_info_ptr));
    CUDA_CHECK(hipMemcpyAsync(&info, device_info_ptr, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    return (info==0);
}